#include "hip/hip_runtime.h"
/*
* Contar cuerpos celestes
*
* Asignatura Computación Paralela (Grado Ingeniería Informática)
* Código secuencial base
*
* @author Ana Moretón Fernández, Arturo Gonzalez-Escribano
* @version v1.3
*
* (c) 2017, Grupo Trasgo, Universidad de Valladolid
*/

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cputils.h"

/* Substituir min por el operador */
#define min(x,y)    ((x) < (y)? (x) : (y))

// Definición de constantes
#define currentGPU 0
#define BLOCK_DIM_FILAS 128
#define BLOCK_DIM_COLUMNAS 8
#define MAX_THREADS 1024

__global__ void etiquetadoInicialKernel(int* matrixDataDev, int* matrixResultDev, int* matrixResultCopyDev, int i, int columns){

	/*Identificaciones necesarios*/
	int IDX_Thread		=	threadIdx.x;				//Identificacion del hilo en la dimension x
	int IDY_Thread		=	threadIdx.y;				//Identificacion del hilo en la dimension y

	int IDX_block		=	blockIdx.x;				//Identificacion del bloque en la dimension x
	int IDY_block		=	blockIdx.y;				//Identificacion del bloque en la dimension y

	int shapeBlock_X	=	blockDim.x;				//Numeros del bloques en la dimension x
	int shapeBlock_Y	=	blockDim.y;				//Numeros del bloques en la dimension y

	

	matrixResultCopyDev[i*(columns)+j]=-1;
	matrixResultDev[i*(columns)+j]=-1;
	// Si es 0 se trata del fondo y no lo computamos
	if(matrixDataDev[i*(columns)+j]!=0){
		matrixResultDev[i*(columns)+j]=i*(columns)+j;
	}
}



/**
* Funcion secuencial para la busqueda de mi bloque
*/
int computation(int x, int y, int columns, int* matrixData, int *matrixResult, int *matrixResultCopy){
	// Inicialmente cojo mi indice
	int result=matrixResultCopy[x*columns+y];
	if( result!= -1){
		//Si es de mi mismo grupo, entonces actualizo
		if(matrixData[(x-1)*columns+y] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[(x-1)*columns+y]);
		}
		if(matrixData[(x+1)*columns+y] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[(x+1)*columns+y]);
		}
		if(matrixData[x*columns+y-1] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[x*columns+y-1]);
		}
		if(matrixData[x*columns+y+1] == matrixData[x*columns+y])
		{
			result = min (result, matrixResultCopy[x*columns+y+1]);
		}

		// Si el indice no ha cambiado retorna 0
		if(matrixResult[x*columns+y] == result){ return 0; }
		// Si el indice cambia, actualizo matrix de resultados con el indice adecuado y retorno 1
		else { matrixResult[x*columns+y]=result; return 1;}

	}
	return 0;
}

/**
* Funcion principal
*/
int main (int argc, char* argv[])
{

	/* 1. Leer argumento y declaraciones */
	if (argc < 2) 	{
		printf("Uso: %s <imagen_a_procesar>\n", argv[0]);
		return(EXIT_SUCCESS);
	}
	char* image_filename = argv[1];

	int rows=-1;
	int columns =-1;
	int *matrixData=NULL, *matrixDataDev=NULL;
	int *matrixResult=NULL, *matrixResultDev=NULL;
	int *matrixResultCopy=NULL, *matrixResultCopyDev=NULL;
	int numBlocks=-1;



	/* 2. Leer Fichero de entrada e inicializar datos */

	/* 2.1 Abrir fichero */
	FILE *f = cp_abrir_fichero(image_filename);

	// Compruebo que no ha habido errores
	if (f==NULL)
	{
	   perror ("Error al abrir fichero.txt");
	   return -1;
	}

	/* 2.2 Leo valores del fichero */
	int i,j;
	fscanf (f, "%d\n", &rows);
	fscanf (f, "%d\n", &columns);
	// Añado dos filas y dos columnas mas para los bordes
	rows=rows+2;
	columns = columns+2;

	/* 2.3 Reservo la memoria necesaria para la matriz de datos */
	matrixData= (int *)malloc( rows*(columns) * sizeof(int) );
	if ( (matrixData == NULL)   ) {
 		perror ("Error reservando memoria");
	   	return -1;
	}

	/* 2.4 Inicializo matrices */
	for(i=0;i< rows; i++){
		for(j=0;j< columns; j++){
			matrixData[i*(columns)+j]=-1;
		}
	}
	/* 2.5 Relleno bordes de la matriz */
	for(i=1;i<rows-1;i++){
		matrixData[i*(columns)+0]=0;
		matrixData[i*(columns)+columns-1]=0;
	}
	for(i=1;i<columns-1;i++){
		matrixData[0*(columns)+i]=0;
		matrixData[(rows-1)*(columns)+i]=0;
	}
	/* 2.6 Relleno la matriz con los datos del fichero */
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			fscanf (f, "%d\n", &matrixData[i*(columns)+j]);
		}
	}
	fclose(f);

	#ifdef WRITE
		printf("Inicializacion \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixData[i*(columns)+j]);
			}
			printf("\n");
		}
	#endif

	hipSetDevice(0);
	hipDeviceSynchronize();

	/* PUNTO DE INICIO MEDIDA DE TIEMPO */
	double t_ini = cp_Wtime();

//
// EL CODIGO A PARALELIZAR COMIENZA AQUI
//

	/* 3. Etiquetado inicial */

	error = hipMalloc(&matrixDataDev, rows*columns* sizeof(int));
	if(error != hipSuccess){
		printf("error en maloc matrixDataDev");
	}
  hipMemcpy(matrixData, matrixDataDev, rows*columns* sizeof(int), hipMemcpyHostToDevice);
  matrixResult= (int *)malloc( (rows)*(columns) * sizeof(int));
  hipMalloc(&matrixResultDev, rows*columns* sizeof(int));
  matrixResultCopy= (int *)malloc( (rows)*(columns) * sizeof(int) );
  hipMalloc(&matrixResultCopyDev, rows*columns* sizeof(int));
  if ( (matrixResult == NULL)  || (matrixResultCopy == NULL)  ) {
     perror ("Error reservando memoria");
       return -1;
  }
	dim3 block(BLOCK_DIM_COLUMNAS,BLOCK_DIM_FILAS);
	int num_col_grid, num_fil_grid;
	if( columns % BLOCK_DIM_COLUMNAS != 0)
		num_col_grid = columns/BLOCK_DIM_COLUMNAS+1;
	else
		num_col_grid = columns/BLOCK_DIM_COLUMNAS;
	if( rows % BLOCK_DIM_FILAS != 0)
		num_fil_grid = rows/BLOCK_DIM_FILAS+1;
	else
		num_fil_grid = rows/BLOCK_DIM_FILAS;

	//printf("F%d-C%d\n", num_fil_grid, num_col_grid);
dim3 grid(num_col_grid, num_fil_grid);

numBlocks = num_col_grid * num_fil_grid;
// printf ("%d\n",numBlocks);


		etiquetadoInicialKernel<<< grid, block>>>(matrixDataDev, matrixResultDev, matrixResultCopyDev, rows, columns);
	// return 0;

	/* 4. Computacion */
	int t=0;
	/* 4.1 Flag para ver si ha habido cambios y si se continua la ejecucion */
	int flagCambio=1;

	/* 4.2 Busqueda de los bloques similiares */
	for(t=0; flagCambio !=0; t++){
		flagCambio=0;

		/* 4.2.1 Actualizacion copia */
		for(i=1;i<rows-1;i++){
			for(j=1;j<columns-1;j++){
				if(matrixResult[i*(columns)+j]!=-1){
					matrixResultCopy[i*(columns)+j]=matrixResult[i*(columns)+j];
				}
			}
		}

		/* 4.2.2 Computo y detecto si ha habido cambios */
		for(i=1;i<rows-1;i++){
			for(j=1;j<columns-1;j++){
				flagCambio= flagCambio+ computation(i,j,columns, matrixData, matrixResult, matrixResultCopy);
			}
		}

		#ifdef DEBUG
			printf("\nResultados iter %d: \n", t);
			for(i=0;i<rows;i++){
				for(j=0;j<columns;j++){
					printf ("%d\t", matrixResult[i*columns+j]);
				}
				printf("\n");
			}
		#endif

	}

	/* 4.3 Inicio cuenta del numero de bloques */
	numBlocks=0;
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			if(matrixResult[i*columns+j] == i*columns+j) numBlocks++;
		}
	}

//
// EL CODIGO A PARALELIZAR TERMINA AQUI
//

	/* PUNTO DE FINAL DE MEDIDA DE TIEMPO */
	hipDeviceSynchronize();
 	double t_fin = cp_Wtime();


	/* 5. Comprobación de resultados */
  	double t_total = (double)(t_fin - t_ini);

	printf("Result: %d:%d\n", numBlocks, t);
	printf("Time: %lf\n", t_total);
	#ifdef WRITE
		printf("Resultado: \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixResult[i*columns+j]);
			}
			printf("\n");
		}
	#endif

	/* 6. Liberacion de memoria */
	free(matrixData);
	free(matrixResult);
	free(matrixResultCopy);

	return 0;
}
